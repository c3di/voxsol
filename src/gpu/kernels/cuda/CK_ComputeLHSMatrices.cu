#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>

#define cudaCheckSuccess(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__
void cuda_ComputeLHSMatrices(REAL* integrals_linear ) {

}
