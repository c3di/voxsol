#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include "gpu/CudaCommonFunctions.h"


__global__
void cuda_SolveDisplacement(REAL* d_displacements, unsigned short* d_signatureIds, REAL* d_fragmentSignatures) {
    int idx = threadIdx.x*3;
    int sigId = d_signatureIds[threadIdx.x];
    int sigIdx = sigId * 27 * 9;

    d_displacements[idx    ] = sigId;
    d_displacements[idx + 1] = d_fragmentSignatures[sigIdx];
    d_displacements[idx + 2] = d_fragmentSignatures[sigIdx+1];
}

extern "C" void CK_SolveDisplacement_launch(REAL* d_displacements, unsigned short* d_signatureIds, REAL* d_fragmentSignatures, unsigned int numVertices) {

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(64, 1, 1);
    
    cuda_SolveDisplacement <<< grid, threads >>>(d_displacements, d_signatureIds, d_fragmentSignatures);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Cuda launch failed: %s", hipGetErrorString(err));
    }
}
