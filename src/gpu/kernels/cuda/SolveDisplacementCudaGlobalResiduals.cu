#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>
#include <>
#include "gpu/CudaCommonFunctions.h"
#include "solution/Vertex.h"
#include "solution/samplers/BlockSampler.h"


#define MATRIX_ENTRY(rhsMatricesStartPointer, matrixIndex, row, col) rhsMatricesStartPointer[matrixIndex*9 + col*3 + row]

#define LHS_MATRIX_INDEX 13            // Position of the LHS matrix in the material config equations
#define EQUATION_ENTRY_SIZE 9 * 27 + 3 // 27 3x3 matrices and one 1x3 vector for Neumann stress
#define NEUMANN_OFFSET 9 * 27          // Offset to the start of the Neumann stress vector inside an equation block
#define UPDATES_PER_THREAD 100  // Number of vertices that should be updated stochastically per thread per kernel execution

//#define DYN_ADJUSTMENT_MAX 0.01f

__device__
int getGlobalIdx_1D_3DGlobal() {
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x
        + threadIdx.y * blockDim.x + threadIdx.x;
}

__device__
int getGlobalIdx_3D_3DGlobal() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__device__ bool isInsideSolution(int offsetX, int offsetY, int offsetZ, const uint3 solutionDimensions) {
    return offsetX >= 0 && offsetX < solutionDimensions.x && offsetY >= 0 && offsetY < solutionDimensions.y && offsetZ >= 0 && offsetZ < solutionDimensions.z;
}

__device__ void buildRHSVectorForVertexGlobal(
    REAL* rhsVec,
    Vertex* verticesOnGPU,
    const REAL* matrices,
    short centerCoordX,
    short centerCoordY,
    short centerCoordZ,
    const uint3 solutionDimensions
) {
    int localNeighborIndex = 0;
    unsigned int globalNeighborIndex = 0;
    Vertex dummy;

    // Build RHS vector by multiplying each neighbor's displacement with its RHS matrix
    for (char localOffsetZ = 0; localOffsetZ <= 2; localOffsetZ++) {
        for (char localOffsetY = 0; localOffsetY <= 2; localOffsetY++) {
            for (char localOffsetX = 0; localOffsetX <= 2; localOffsetX++) {

                if (localOffsetZ == 1 && localOffsetY == 1 && localOffsetX == 1) {
                    //This is the center vertex that we're solving for, so skip it
                    continue;
                }

                unsigned short localNeighborCoordX = centerCoordX + localOffsetX - 1;
                unsigned short localNeighborCoordY = centerCoordY + localOffsetY - 1;
                unsigned short localNeighborCoordZ = centerCoordZ + localOffsetZ - 1;

                //Local problem size is always 3x3x3 vertices, regardless of solution size
                localNeighborIndex = localOffsetZ * 9 + localOffsetY * 3 + localOffsetX;
                globalNeighborIndex = solutionDimensions.y*solutionDimensions.x*localNeighborCoordZ + solutionDimensions.x*localNeighborCoordY + localNeighborCoordX;

                Vertex neighbor;
                if (isInsideSolution(localNeighborCoordX, localNeighborCoordY, localNeighborCoordZ, solutionDimensions)) {
                    neighbor = verticesOnGPU[globalNeighborIndex];
                }
                else {
                    neighbor = dummy;
                }

                // RHS[neighbor] * displacement[neighbor]
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 0) * neighbor.x;
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 1) * neighbor.y;
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 2) * neighbor.z;

                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 0) * neighbor.x;
                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 1) * neighbor.y;
                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 2) * neighbor.z;

                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 0) * neighbor.x;
                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 1) * neighbor.y;
                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 2) * neighbor.z;
                
            }
        }
    }
}

__device__ const REAL* getPointerToMatricesForVertexGlobal(Vertex& vertex, const REAL* matConfigEquations) {
    unsigned int equationIndex = static_cast<unsigned int>(vertex.materialConfigId) * (EQUATION_ENTRY_SIZE);
    return &matConfigEquations[equationIndex];
}

__device__ void updateVertexGlobalResidual(Vertex& vertexToUpdate, REAL* rhsVec, const REAL* matrices) {

    //Move to right side of equation and apply Neumann stress
    rhsVec[0] = -rhsVec[0] + matrices[NEUMANN_OFFSET];
    rhsVec[1] = -rhsVec[1] + matrices[NEUMANN_OFFSET + 1];
    rhsVec[2] = -rhsVec[2] + matrices[NEUMANN_OFFSET + 2];

    //rhsVec * LHS^-1
    REAL dx = MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 0) * rhsVec[0] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 0) * rhsVec[1] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 0) * rhsVec[2];
    REAL dy = MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 1) * rhsVec[0] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 1) * rhsVec[1] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 1) * rhsVec[2];
    REAL dz = MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 2) * rhsVec[0] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 2) * rhsVec[1] +
        MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 2) * rhsVec[2];

#ifdef DYN_ADJUSTMENT_MAX
    REAL diff = (abs(vertexToUpdate.x - dx) + abs(vertexToUpdate.y - dy) + abs(vertexToUpdate.z - dz)) * 0.3333333f;
    if (diff > DYN_ADJUSTMENT_MAX) {
        // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
        // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
        // of surrounding vertices
        printf("Bad adjustment: %f diff for %f,%f,%f \n", diff, rhsVec[0], rhsVec[1], rhsVec[2]);
        return;
    }
#endif

    vertexToUpdate.x = dx;
    vertexToUpdate.y = dy;
    vertexToUpdate.z = dz;
}

__device__ void addResidualFromFullresVertex(
    unsigned int fullresX,
    unsigned int fullresY,
    unsigned int fullresZ,
    REAL* residual,
    const REAL* matConfigEquations,
    const uint3 solutionDimensions,
    Vertex* verticesOnGPU
) {
    int fullresIndex = solutionDimensions.y*solutionDimensions.x*fullresZ + solutionDimensions.x*fullresY + fullresX;
    if (!isInsideSolution(fullresX, fullresY, fullresZ, solutionDimensions)) {
        // this vertex could lie outside the solution space because we expand the working block by 1 when gathering residuals, in this case residual is 0
        return;
    }
    Vertex globalFullresVertex = verticesOnGPU[fullresIndex];
    if (globalFullresVertex.materialConfigId == static_cast<ConfigId>(0)) {
        // config id 0 should always be the case where the vertex is surrounded by empty cells, therefore not updateable so residual is 0
        return;
    }
    REAL oldX = globalFullresVertex.x;
    REAL oldY = globalFullresVertex.y;
    REAL oldZ = globalFullresVertex.z;
    REAL rhsVec[3] = { 0,0,0 };
    const REAL* matrices = getPointerToMatricesForVertexGlobal(globalFullresVertex, matConfigEquations);
    buildRHSVectorForVertexGlobal(rhsVec, verticesOnGPU, matrices, fullresX, fullresY, fullresZ, solutionDimensions);
    updateVertexGlobalResidual(globalFullresVertex, rhsVec, matrices);

    // Get the magnitude of the displacement difference (residual)
    oldX = globalFullresVertex.x - oldX;
    oldY = globalFullresVertex.y - oldY;
    oldZ = globalFullresVertex.z - oldZ;
    oldX *= oldX;
    oldY *= oldY;
    oldZ *= oldZ;
    *residual += oldX + oldY + oldZ;
}

__device__ void updateResidualsLevelZeroGlobal(
    Vertex* verticesOnGPU,
    REAL* importanceVolume,
    const REAL* matConfigEquations,
    const uint3& blockOriginCoord,
    const uint3 solutionDimensions
) {
    // We want to find residuals for vertices bordering our BLOCK_SIZE area too, so -1, then project to level 0 with / 2
    unsigned int levelZeroX = (blockOriginCoord.x > 0 ? blockOriginCoord.x - 1 : 0) / 2;
    unsigned int levelZeroY = (blockOriginCoord.y > 0 ? blockOriginCoord.y - 1 : 0) / 2;
    unsigned int levelZeroZ = (blockOriginCoord.z > 0 ? blockOriginCoord.z - 1 : 0) / 2;

    unsigned int vertexToUpdateX = levelZeroX + threadIdx.x;
    unsigned int vertexToUpdateY = levelZeroY + threadIdx.y;
    unsigned int vertexToUpdateZ = levelZeroZ + threadIdx.z;

    unsigned int updateRange = (BLOCK_SIZE + 2) / 2;

    if (vertexToUpdateX > levelZeroX + updateRange ||
        vertexToUpdateY > levelZeroY + updateRange ||
        vertexToUpdateZ > levelZeroZ + updateRange)
    {
        // Since level 0 has half the vertices some threads may be unnecessary
        return;
    }

    // Precompute the index of the residual we want to update on Level 0
    unsigned int residualIndex = vertexToUpdateZ * solutionDimensions.x / 2 * solutionDimensions.y / 2;
    residualIndex += vertexToUpdateY * solutionDimensions.x / 2;
    residualIndex += vertexToUpdateX;

    REAL residual = asREAL(0.0);

    // Project back down to fullres
    vertexToUpdateX *= 2;
    vertexToUpdateY *= 2;
    vertexToUpdateZ *= 2;

    // Pool the residuals from the fullres level that contribute to this Level 0 vertex's residual
    addResidualFromFullresVertex(vertexToUpdateX, vertexToUpdateY, vertexToUpdateZ, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX + 1, vertexToUpdateY, vertexToUpdateZ, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX, vertexToUpdateY + 1, vertexToUpdateZ, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX + 1, vertexToUpdateY + 1, vertexToUpdateZ, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);

    addResidualFromFullresVertex(vertexToUpdateX, vertexToUpdateY, vertexToUpdateZ + 1, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX + 1, vertexToUpdateY, vertexToUpdateZ + 1, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX, vertexToUpdateY + 1, vertexToUpdateZ + 1, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);
    addResidualFromFullresVertex(vertexToUpdateX + 1, vertexToUpdateY + 1, vertexToUpdateZ + 1, &residual, matConfigEquations, solutionDimensions, verticesOnGPU);

    importanceVolume[residualIndex] = residual;
}

__device__ void updateVerticesStochasticallyGlobalResiduals(
    Vertex* verticesOnGPU,
    const REAL* matConfigEquations,
    hiprandState localRNGState,
    const uint3& blockOriginCoord,
    const uint3 solutionDimensions
) {

    for (int i = 0; i < UPDATES_PER_THREAD; i++) {
        int offsetX = blockOriginCoord.x + lroundf(hiprand_uniform(&localRNGState) * BLOCK_SIZE);
        int offsetY = blockOriginCoord.y + lroundf(hiprand_uniform(&localRNGState) * BLOCK_SIZE);
        int offsetZ = blockOriginCoord.z + lroundf(hiprand_uniform(&localRNGState) * BLOCK_SIZE);
        if (!isInsideSolution(offsetX, offsetY, offsetZ, solutionDimensions)) {
            continue;
        }
        int offset = solutionDimensions.y*solutionDimensions.x*offsetZ + solutionDimensions.x*offsetY + offsetX;
        Vertex globalVertexToUpdate = verticesOnGPU[offset];
        if (globalVertexToUpdate.materialConfigId == static_cast<ConfigId>(0)) {
            // config id 0 should always be the case where the vertex is surrounded by empty cells, therefore not updateable
            continue;
        }
        const REAL* matrices = getPointerToMatricesForVertexGlobal(globalVertexToUpdate, matConfigEquations);
        REAL rhsVec[3] = { 0,0,0 };

        buildRHSVectorForVertexGlobal(rhsVec, verticesOnGPU, matrices, offsetX, offsetY, offsetZ, solutionDimensions);
        updateVertexGlobalResidual(globalVertexToUpdate, rhsVec, matrices);

        verticesOnGPU[offset] = globalVertexToUpdate;
    }

}

__global__
void cuda_SolveDisplacementGlobalResiduals(
    Vertex* verticesOnGPU,
    REAL* matConfigEquations,
    REAL* importanceVolume,
    const uint3 solutionDimensions,
    hiprandState* globalRNGStates,
    const uint3* blockOrigins
) {
    const uint3 blockOriginCoord = blockOrigins[blockIdx.x];
    if (blockOriginCoord.x >= solutionDimensions.x || blockOriginCoord.y >= solutionDimensions.y || blockOriginCoord.z >= solutionDimensions.z) {
        // Some blocks may have been set to an invalid value during the importance sampling phase if they overlap with some other block, these
        // should not be processed
        return;
    }
    hiprandState localRNGState = globalRNGStates[getGlobalIdx_1D_3DGlobal()];

    updateVerticesStochasticallyGlobalResiduals(verticesOnGPU, matConfigEquations, localRNGState, blockOriginCoord, solutionDimensions);

    __syncthreads();

    updateResidualsLevelZeroGlobal(verticesOnGPU, importanceVolume, matConfigEquations, blockOriginCoord, solutionDimensions);
}

__global__
void cuda_init_curand_stateGlobal(hiprandState* rngState) {
    int id = getGlobalIdx_1D_3DGlobal();
    // seed, sequence number, offset, hiprandState
    hiprand_init(clock64(), id, 0, &rngState[id]);
}

__host__
extern "C" void cudaInitializeRNGStatesGlobal(hiprandState** rngStateOnGPU) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // setup execution parameters
    dim3 threadsPerBlock(BLOCK_SIZE - 1, BLOCK_SIZE - 1, BLOCK_SIZE - 1);
    int maxConcurrentBlocks = deviceProperties.multiProcessorCount * 4; //TODO: Calculate this based on GPU max for # blocks
    int numThreads = maxConcurrentBlocks * threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z;

    cudaCheckSuccess(hipMalloc(rngStateOnGPU, sizeof(hiprandState) * numThreads));
    cuda_init_curand_stateGlobal << < maxConcurrentBlocks, threadsPerBlock >> > (*rngStateOnGPU);
    hipDeviceSynchronize();
    cudaCheckExecution();
}

__host__
extern "C" void cudaLaunchSolveDisplacementKernelGlobalResiduals(
    Vertex* vertices,
    REAL* matConfigEquations,
    REAL* importanceVolume,
    hiprandState* rngStateOnGPU,
    uint3* blockOrigins,
    const int numBlockOrigins,
    const uint3 solutionDims
) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // setup execution parameters
    dim3 threadsPerBlock(BLOCK_SIZE - 1, BLOCK_SIZE - 1, BLOCK_SIZE - 1);
    int maxConcurrentBlocks = deviceProperties.multiProcessorCount * 4; //TODO: Calculate this based on GPU max for # blocks
    int numIterations = std::max(numBlockOrigins / maxConcurrentBlocks, 1);
    
    for (int i = 0; i < numIterations; i++) {
        uint3* currentBlockOrigins = &blockOrigins[i * maxConcurrentBlocks];
        int numBlocks = std::min(numBlockOrigins - i*maxConcurrentBlocks, maxConcurrentBlocks);
        cuda_SolveDisplacementGlobalResiduals << < numBlocks, threadsPerBlock >> >(vertices, matConfigEquations, importanceVolume, solutionDims, rngStateOnGPU, currentBlockOrigins);
        hipDeviceSynchronize();
        cudaCheckExecution();
    }
}


