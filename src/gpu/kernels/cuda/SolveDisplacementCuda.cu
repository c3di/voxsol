#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>
#include <>
#include "gpu/CudaCommonFunctions.h"
#include "solution/Vertex.h"


#define MATRIX_ENTRY(rhsMatricesStartPointer, matrixIndex, row, col) rhsMatricesStartPointer[matrixIndex*9 + col*3 + row]

#define LHS_MATRIX_INDEX 13            // Position of the LHS matrix in the material config equations
#define EQUATION_ENTRY_SIZE 9 * 27 + 3 // 27 3x3 matrices and one 1x3 vector for Neumann stress
#define BLOCK_SIZE 6                   // Number of threads in one block dimension (total threads per block is BLOCK_SIZE^3)
#define WORKING_AREA_SIZE 4            // Vertices that are 'active', minus the 1 vertex border of reference vertices that aren't updated in the kernel
#define NEUMANN_OFFSET 9 * 27          // Offset to the start of the Neumann stress vector inside an equation block
#define UPDATES_PER_THREAD 50          // Number of vertices that should be updated stochastically per thread per kernel execution

__device__
int getGlobalIdx_1D_3D() {
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x
        + threadIdx.y * blockDim.x + threadIdx.x;
}

__device__
int getGlobalIdx_3D_3D() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__device__ void buildRHSVectorForVertex(REAL* rhsVec, Vertex localVertices[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE], REAL* matrices, short centerCoordX, short centerCoordY, short centerCoordZ) {
    int localNeighborIndex = 0;

    // Build RHS vector by multiplying each neighbor's displacement with its RHS matrix
    for (char localOffsetZ = 0; localOffsetZ <= 2; localOffsetZ++) {
        for (char localOffsetY = 0; localOffsetY <= 2; localOffsetY++) {
            for (char localOffsetX = 0; localOffsetX <= 2; localOffsetX++) {

                if (localOffsetZ == 1 && localOffsetY == 1 && localOffsetX == 1) {
                    //This is the center vertex that we're solving for, so skip it
                    continue;
                }

                unsigned short localNeighborCoordX = centerCoordX + localOffsetX - 1;
                unsigned short localNeighborCoordY = centerCoordY + localOffsetY - 1;
                unsigned short localNeighborCoordZ = centerCoordZ + localOffsetZ - 1;

                //Local problem size is always 3x3x3 vertices, regardless of solution size
                localNeighborIndex = localOffsetZ * 9 + localOffsetY * 3 + localOffsetX;

                const Vertex* neighbor = &localVertices[localNeighborCoordX][localNeighborCoordY][localNeighborCoordZ];

                // RHS[neighbor] * displacement[neighbor]
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 0) * neighbor->x;
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 1) * neighbor->y;
                rhsVec[0] += MATRIX_ENTRY(matrices, localNeighborIndex, 0, 2) * neighbor->z;

                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 0) * neighbor->x;
                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 1) * neighbor->y;
                rhsVec[1] += MATRIX_ENTRY(matrices, localNeighborIndex, 1, 2) * neighbor->z;

                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 0) * neighbor->x;
                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 1) * neighbor->y;
                rhsVec[2] += MATRIX_ENTRY(matrices, localNeighborIndex, 2, 2) * neighbor->z;
            }
        }
    }
}

__device__ REAL* getPointerToMatricesForVertex(Vertex* vertex, REAL* matConfigEquations) {
    short equationId = vertex->materialConfigId;
    short equationIndex = equationId * (EQUATION_ENTRY_SIZE);
    return &matConfigEquations[equationIndex];
}

// This function is called from inside a conditional, do not place any __syncthreads() in here!
__device__ void updateVerticesStochastically(Vertex localVertices[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE], REAL* matConfigEquations, hiprandState localRNGState ) {

    for (int i = 0; i < UPDATES_PER_THREAD; i++) {
        // There's a 1 vertex border around the problem area that shouldn't be updated, so choose something in the middle region
        short offsetX = 1 + lroundf(hiprand_uniform(&localRNGState) * (WORKING_AREA_SIZE - 1)); 
        short offsetY = 1 + lroundf(hiprand_uniform(&localRNGState) * (WORKING_AREA_SIZE - 1));
        short offsetZ = 1 + lroundf(hiprand_uniform(&localRNGState) * (WORKING_AREA_SIZE - 1));

        Vertex* localVertexToUpdate = &localVertices[offsetX][offsetY][offsetZ];
        REAL* matrices = getPointerToMatricesForVertex(localVertexToUpdate, matConfigEquations);
        REAL rhsVec[3] = { 0,0,0 };
        
        buildRHSVectorForVertex(rhsVec, localVertices, matrices, offsetX, offsetY, offsetZ);

        //Move to right side of equation and apply Neumann stress
        rhsVec[0] = -rhsVec[0] + matrices[NEUMANN_OFFSET];
        rhsVec[1] = -rhsVec[1] + matrices[NEUMANN_OFFSET + 1];
        rhsVec[2] = -rhsVec[2] + matrices[NEUMANN_OFFSET + 2];

        //rhsVec * LHS^-1
        localVertexToUpdate->x =
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 0) * rhsVec[0] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 0) * rhsVec[1] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 0) * rhsVec[2];

        localVertexToUpdate->y =
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 1) * rhsVec[0] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 1) * rhsVec[1] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 1) * rhsVec[2];

        localVertexToUpdate->z =
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, 2) * rhsVec[0] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, 2) * rhsVec[1] +
            MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, 2) * rhsVec[2];

    }
}

__global__
void cuda_SolveDisplacement(Vertex* verticesOnGPU, REAL* matConfigEquations, const SolutionDim solutionDimensions, hiprandState* globalRNGStates, int3* blockOrigins) {
    // Dummy vertex is used for any vertex that lies outside the solution. MatID is designed to cause an exception if one of these vertices is actually worked on
    Vertex dummyVertex;
    dummyVertex.materialConfigId = 999;
    int3* blockOriginCoord = blockOrigins + blockIdx.x;
    hiprandState localRNGState = globalRNGStates[getGlobalIdx_1D_3D()];

    __shared__ Vertex localVertices[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];

    short threadVertexX = blockOriginCoord->x + threadIdx.x;
    short threadVertexY = blockOriginCoord->y + threadIdx.y;
    short threadVertexZ = blockOriginCoord->z + threadIdx.z;
    int threadVertexIndex = threadVertexZ * solutionDimensions.x * solutionDimensions.y + threadVertexY * solutionDimensions.x + threadVertexX;

    bool isInsideSolutionSpace = threadVertexX >= 0 && threadVertexX < solutionDimensions.x && 
        threadVertexY >= 0 && threadVertexY < solutionDimensions.y && 
        threadVertexZ >= 0 && threadVertexZ < solutionDimensions.z;

    if (isInsideSolutionSpace) {
        localVertices[threadIdx.x][threadIdx.y][threadIdx.z] = verticesOnGPU[threadVertexIndex];
    } else {
        localVertices[threadIdx.x][threadIdx.y][threadIdx.z] = dummyVertex;
    }

    //__syncthreads must be called outside of any conditional code, but it must be called before going on to ensure shared memory has been initialized
    __syncthreads();

    if (isInsideSolutionSpace) {
        updateVerticesStochastically(localVertices, matConfigEquations, localRNGState);
        verticesOnGPU[threadVertexIndex] = localVertices[threadIdx.x][threadIdx.y][threadIdx.z];
    } else {
         // This thread was responsible for setting one of the border vertices that are not being updated, they're only there to provide input for the neighboring active
         // vertices. After transferring this data to shared memory this thread is finished.
    }
}

__global__
void cuda_init_curand_state(hiprandState* rngState) {
    int id = getGlobalIdx_3D_3D();
    // seed, sequence number, offset, hiprandState
    hiprand_init(id, 0, 0, &rngState[id]);
}

__host__
hiprandState* initializeRNGStates(int numConcurrentBlocks, dim3 threadsPerBlock) {
    int numThreads = numConcurrentBlocks * BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE;
    hiprandState* rngStateOnGPU;
    cudaCheckSuccess(hipMalloc(&rngStateOnGPU, sizeof(hiprandState) * numThreads));
    cuda_init_curand_state<<< numConcurrentBlocks, threadsPerBlock >>> (rngStateOnGPU);
    cudaCheckExecution();
    return rngStateOnGPU;
}

__host__
void generateBlockOrigins(int3* blockOrigins, int numConcurrentBlocks, const SolutionDim solutionDims) {
    std::random_device rd;
    std::mt19937 rng(rd());
    // Choose a vertex as the origin (bottom left corner) of a block. Starts at -1 because blocks have a 1-vertex 
    // border of fixed, zero displacement 'virtual' vertices
    std::uniform_int_distribution<int> distX(-1, std::max(solutionDims.x - WORKING_AREA_SIZE - 1, -1));
    std::uniform_int_distribution<int> distY(-1, std::max(solutionDims.y - WORKING_AREA_SIZE - 1, -1));
    std::uniform_int_distribution<int> distZ(-1, std::max(solutionDims.z - WORKING_AREA_SIZE - 1, -1));

    for (int b = 0; b < numConcurrentBlocks; b++) {
        blockOrigins[b].x = distX(rng);
        blockOrigins[b].y = distY(rng);
        blockOrigins[b].z = distZ(rng);
    }
}

__host__
extern "C" void cudaLaunchSolveDisplacementKernel(Vertex* vertices, REAL* matConfigEquations, const SolutionDim solutionDims) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // setup execution parameters
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    int numConcurrentBlocks = deviceProperties.multiProcessorCount;

    // setup hiprand
    hiprandState* rngStateOnGPU = initializeRNGStates(numConcurrentBlocks, threadsPerBlock);
    hipDeviceSynchronize();
   
    int3* blockOrigins;
    cudaCheckSuccess(hipMallocManaged(&blockOrigins, sizeof(int3) * numConcurrentBlocks));

    for (int i = 0; i < 100; i++) {
        generateBlockOrigins(blockOrigins, numConcurrentBlocks, solutionDims);
        cuda_SolveDisplacement <<< numConcurrentBlocks, threadsPerBlock >>>(vertices, matConfigEquations, solutionDims, rngStateOnGPU, blockOrigins);
        cudaCheckExecution();
        hipDeviceSynchronize();
    }

    cudaCheckSuccess(hipFree(blockOrigins));
    cudaCheckSuccess(hipFree(rngStateOnGPU));
}


