#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>
#include <>
#include "solution/Vertex.h"
#include "solution/samplers/BlockSampler.h"
#include "gpu/sampling/ResidualVolume.h"
#include "gpu/CudaCommonFunctions.h"

#define MATRIX_ENTRY(rhsMatricesStartPointer, matrixIndex, row, col) rhsMatricesStartPointer[matrixIndex*9 + col*3 + row]

#define LHS_MATRIX_INDEX        13          // Position of the LHS matrix in the material config equations
#define EQUATION_ENTRY_SIZE     9 * 27 + 3  // 27 3x3 matrices and one 1x3 vector for Neumann stress
#define NEUMANN_OFFSET          9 * 27      // Offset to the start of the Neumann stress vector inside an equation block
#define UPDATES_PER_THREAD      200          // Number of vertices that should be updated stochastically per worker 
#define NUM_WORKERS             6           // Number of workers that are updating vertices in parallel (one warp per worker)

//#define DYN_ADJUSTMENT_MAX 0.01f

__device__
int getGlobalIdx_1D_3DGlobal() {
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x
        + threadIdx.y * blockDim.x + threadIdx.x;
}

__device__
int getGlobalIdx_3D_3DGlobal() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__device__ bool isInsideSolution(const uint3 coord, const uint3 solutionDimensions) {
    return coord.x < solutionDimensions.x && coord.y < solutionDimensions.y && coord.z < solutionDimensions.z;
}

__device__ void buildRHSVectorForVertex(
    REAL rhsVec[NUM_WORKERS][3],
    Vertex localVertices[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2],
    const REAL* matrices,
    const char3& localCenterCoord
) {
    // We want to keep a full warp dedicated to each worker, but we only need enough threads for the 27 neighbors (minus the center vertex)
    const bool activeThread = threadIdx.x < 27 && threadIdx.x != 13;

    REAL rhsEntry = 0;
    unsigned mask = __ballot_sync(0xffffffff, activeThread);

    if (activeThread) {
        // Get coords of neighbor that this thread is responsible for, relative to the center vertex, in the 3x3x3 local problem
        char3 localNeighborCoord;
        localNeighborCoord.z = (localCenterCoord.z + threadIdx.x / 9) - 1;
        localNeighborCoord.y = (localCenterCoord.y + (threadIdx.x / 3) % 3) - 1;
        localNeighborCoord.x = (localCenterCoord.x + threadIdx.x % 3) - 1;

        Vertex* neighbor = &localVertices[localNeighborCoord.z][localNeighborCoord.y][localNeighborCoord.x];

        rhsEntry = MATRIX_ENTRY(matrices, threadIdx.x, threadIdx.y, 0) * neighbor->x;
        rhsEntry += MATRIX_ENTRY(matrices, threadIdx.x, threadIdx.y, 1) * neighbor->y;
        rhsEntry += MATRIX_ENTRY(matrices, threadIdx.x, threadIdx.y, 2) * neighbor->z;

        for (int offset = 16; offset > 0; offset /= 2) {
            rhsEntry += __shfl_down_sync(mask, rhsEntry, offset);
        }

        if (threadIdx.x == 0) {
            // Result of the shuffle reduction is stored in thread 0's variable
            rhsVec[threadIdx.z][threadIdx.y] = rhsEntry;
        }
    } 
}

__device__ const REAL* getPointerToMatricesForVertexGlobal(Vertex* vertex, const REAL* matConfigEquations) {
    unsigned int equationIndex = static_cast<unsigned int>(vertex->materialConfigId) * (EQUATION_ENTRY_SIZE);
    return &matConfigEquations[equationIndex];
}

__device__ void updateVertex(Vertex* vertexToUpdate, REAL rhsVec[NUM_WORKERS][3], const REAL* matrices) {
    // Choose exactly 3 threads in the same warp to sum up the 3 RHS components and solve the system
    if (threadIdx.y == 0 && threadIdx.x < 3) {
        char rhsComponentIndex = threadIdx.x;
        char workerIndex = threadIdx.z;

        // Move to right side of equation and apply Neumann stress
        rhsVec[workerIndex][rhsComponentIndex] = -rhsVec[workerIndex][rhsComponentIndex] + matrices[NEUMANN_OFFSET + rhsComponentIndex];

        __syncwarp();

        REAL newDisplacement = 0;
        newDisplacement += MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 0, rhsComponentIndex) * rhsVec[workerIndex][0];
        newDisplacement += MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 1, rhsComponentIndex) * rhsVec[workerIndex][1];
        newDisplacement += MATRIX_ENTRY(matrices, LHS_MATRIX_INDEX, 2, rhsComponentIndex) * rhsVec[workerIndex][2];

        if (rhsComponentIndex == 0) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->x - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->x = newDisplacement;
        }
        if (rhsComponentIndex == 1) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->y - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->y = newDisplacement;
        }
        if (rhsComponentIndex == 2) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->z - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->z = newDisplacement;
        }

    }
}


__device__ void updateVerticesStochastically(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    const REAL* matConfigEquations,
    hiprandState* localRngState
) {
    __shared__ REAL rhsVec[NUM_WORKERS][3];

    for (int i = 0; i < UPDATES_PER_THREAD; i++) {
        char3 localCoord = { 1,1,1 }; //starts at 1 because 0 is the outer border of fixed vertices, which shouldn't be updated
        localCoord.z += ceilf(hiprand_uniform(localRngState) * BLOCK_SIZE) - 1; //hiprand_uniform is 0.0 exclusive, 1.0 inclusive, shift to 1...n+1 and shift back with -1 to have a true uniform distribution in 0..n
        localCoord.y += ceilf(hiprand_uniform(localRngState) * BLOCK_SIZE) - 1;
        localCoord.x += ceilf(hiprand_uniform(localRngState) * BLOCK_SIZE) - 1;
        Vertex* vertexToUpdate = &localVertices[localCoord.z][localCoord.y][localCoord.x];

        const REAL* matrices = getPointerToMatricesForVertexGlobal(vertexToUpdate, matConfigEquations);

        buildRHSVectorForVertex(rhsVec, localVertices, matrices, localCoord);

        __syncthreads(); // Need to finish reading from shared before we move on to writing, otherwise RHS becomes unstable

        updateVertex(vertexToUpdate, rhsVec, matrices);
    }

}

//#define SLOWER_COPY 1

#ifdef SLOWER_COPY

__device__
void copyVertexFromGlobalToShared(
    volatile Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    volatile Vertex* verticesOnGPU,
    const uint3 blockOriginCoord,
    const uint3 solutionDimensions
) {
    const int blockSizeWithBorder = BLOCK_SIZE + 2;
    const int numThreadsNeeded = blockSizeWithBorder * blockSizeWithBorder; //each thread will copy over a given x,y for all z ("top down")
    const int threadIdx_1D = threadIdx.z * 32 * 3 + threadIdx.y * 32 + threadIdx.x;

    // Choose the first numThreadsNeeded threads to copy over the vertics
    if (threadIdx_1D < numThreadsNeeded) {
        char3 localCoord = { 0,0,0 };
        localCoord.x = 0;
        localCoord.y += threadIdx_1D / blockSizeWithBorder;
        localCoord.z += threadIdx_1D % blockSizeWithBorder;

        for (int x = 0; x < BLOCK_SIZE + 2; x++) {
            localCoord.x = x;
            uint3 globalCoord = { blockOriginCoord.x + localCoord.x - 1, blockOriginCoord.y + localCoord.y - 1, blockOriginCoord.z + localCoord.z - 1 }; //-1 to account for border at both ends
            volatile Vertex* local = &localVertices[localCoord.z][localCoord.y][localCoord.x];

            if (isInsideSolution(globalCoord, solutionDimensions)) {
                int globalIndex = solutionDimensions.y*solutionDimensions.x*globalCoord.z + solutionDimensions.x*globalCoord.y + globalCoord.x;

                //Turns out it's easier to copy the values manually than to get CUDA to play nice with a volatile struct assignment
                volatile Vertex* global = &verticesOnGPU[globalIndex];
                local->x = global->x;
                local->y = global->y;
                local->z = global->z;
                local->materialConfigId = global->materialConfigId;
            }
            else {
                local->x = 0;
                local->y = 0;
                local->z = 0;
                local->materialConfigId = 0;
            }
        }
    }
}

#else

__device__
void copyVertexFromGlobalToShared(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    volatile Vertex* verticesOnGPU,
    const uint3 blockOriginCoord,
    const uint3 solutionDimensions
) {
    const int blockSizeWithBorder = BLOCK_SIZE + 2;
    const int numThreadsNeeded = blockSizeWithBorder * blockSizeWithBorder; //each thread will copy over a given x,y for all z ("top down")
    const int threadIdx_1D = threadIdx.z * 32 * 3 + threadIdx.y * 32 + threadIdx.x;

    // Choose the first numThreadsNeeded threads to copy over the vertics
    if (threadIdx_1D < numThreadsNeeded) {
        char3 localCoord = { 0,0,0 };
        localCoord.x = threadIdx_1D % blockSizeWithBorder;
        localCoord.y = threadIdx_1D / blockSizeWithBorder;
        localCoord.z = 0;

        for (int z = 0; z < BLOCK_SIZE + 2; z++) {
            localCoord.z = z;
            uint3 globalCoord = { blockOriginCoord.x + localCoord.x - 1, blockOriginCoord.y + localCoord.y - 1, blockOriginCoord.z + z - 1 }; //-1 to account for border at both ends
            Vertex* local = &localVertices[localCoord.z][localCoord.y][localCoord.x];

            if (isInsideSolution(globalCoord, solutionDimensions)) {
                int globalIndex = solutionDimensions.y*solutionDimensions.x*globalCoord.z + solutionDimensions.x*globalCoord.y + globalCoord.x;

                //Turns out it's easier to copy the values manually than to get CUDA to play nice with a volatile struct assignment
                volatile Vertex* global = &verticesOnGPU[globalIndex];
                local->x = global->x;
                local->y = global->y;
                local->z = global->z;
                local->materialConfigId = global->materialConfigId;
            }
            else {
                local->x = 0;
                local->y = 0;
                local->z = 0;
                local->materialConfigId = 0;
            }
        }
    }
}

#endif

__device__
void copyVertexFromSharedToGlobal(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    volatile Vertex* verticesOnGPU,
    const uint3 blockOriginCoord,
    const uint3 solutionDimensions
) {
    const int numThreadsNeeded = BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE; //each thread will copy over one vertex in the inner block (without the border)
    const int threadIdx_1D = threadIdx.z * 32 * 3 + threadIdx.y * 32 + threadIdx.x;

    if (threadIdx_1D < numThreadsNeeded) {
        char3 localCoord = {1,1,1};
        localCoord.z += threadIdx_1D / (BLOCK_SIZE * BLOCK_SIZE);
        localCoord.y += (threadIdx_1D / BLOCK_SIZE) % BLOCK_SIZE;
        localCoord.x += threadIdx_1D % BLOCK_SIZE;

        uint3 globalCoord = { 0,0,0 };
        // Move the checkerboard pattern to the start of the block to be updated
        globalCoord.z += localCoord.z + blockOriginCoord.z - 1;
        globalCoord.y += localCoord.y + blockOriginCoord.y - 1;
        globalCoord.x += localCoord.x + blockOriginCoord.x - 1;

        if (isInsideSolution(globalCoord, solutionDimensions)) {
            int globalIndex = solutionDimensions.y*solutionDimensions.x*globalCoord.z + solutionDimensions.x*globalCoord.y + globalCoord.x;
            Vertex* local = &localVertices[localCoord.z][localCoord.y][localCoord.x];
            volatile Vertex* global = &verticesOnGPU[globalIndex];
            global->x = local->x;
            global->y = local->y;
            global->z = local->z;

#ifdef OUTPUT_NAN_DISPLACEMENTS
            if (isnan(localVertices[localCoord.z][localCoord.y][localCoord.x].x)) {
                printf("NAN encountered for block %i coord %u %u %u \n", blockIdx.x, localCoord.x, localCoord.y, localCoord.z);
            }
#endif
        }
    }

    
}

__global__
void cuda_SolveDisplacement(
    volatile Vertex* verticesOnGPU,
    REAL* matConfigEquations,
    REAL* residualVolume,
    const uint3 solutionDimensions,
    const uint3* blockOrigins,
    hiprandState* rngState
) {
    const uint3 blockOriginCoord = blockOrigins[blockIdx.x];
    if (blockOriginCoord.x >= solutionDimensions.x || blockOriginCoord.y >= solutionDimensions.y || blockOriginCoord.z >= solutionDimensions.z) {
        // Some blocks may have been set to an invalid value during the importance sampling phase if they overlap with some other block, these
        // should not be processed
        return;
    }

    __shared__ Vertex localVertices[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
    
    copyVertexFromGlobalToShared(localVertices, verticesOnGPU, blockOriginCoord, solutionDimensions);
    hiprandState localRngState = rngState[blockIdx.x * NUM_WORKERS + threadIdx.z];

    __syncthreads();

    updateVerticesStochastically(localVertices, matConfigEquations, &localRngState);
    
    __syncthreads();

    copyVertexFromSharedToGlobal(localVertices, verticesOnGPU, blockOriginCoord, solutionDimensions); 
    rngState[blockIdx.x * NUM_WORKERS + threadIdx.z] = localRngState;
}

__global__
void cuda_InitCurandState(hiprandState* rngState) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // seed, sequence number, offset, hiprandState
    hiprand_init(43, id, 0, &rngState[id]);
}

__global__
void cuda_invalidateOverlappingBlocks(uint3* candidates, const int numberOfCandidates, const unsigned int updateRegionSize) {
    extern __shared__ uint3 batch[];
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalId >= numberOfCandidates) {
        return;
    }
    int localId = threadIdx.x;
    uint3 myCandidate = candidates[globalId];
    batch[localId] = myCandidate;

    __syncthreads();

    // Walk through the candidates toward the left
    while (localId > 0) {
        localId -= 1;
        uint3 leftNeighbor = batch[localId];
        // Check for cube intersection, if any condition is true the two rectangular regions cannot intersect
        bool doesNotIntersect = false;
        doesNotIntersect = doesNotIntersect || leftNeighbor.x >= myCandidate.x + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.x + updateRegionSize <= myCandidate.x;
        doesNotIntersect = doesNotIntersect || leftNeighbor.z + updateRegionSize <= myCandidate.z;
        doesNotIntersect = doesNotIntersect || leftNeighbor.z >= myCandidate.z + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.y >= myCandidate.y + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.y + updateRegionSize <= myCandidate.y;
        if (!doesNotIntersect) {
            // Invalidate this block, it will later be skipped in the update phase since it lies outside the solution by definition of max_uint
            //printf("Invalid block: %i %i %i with %i %i %i\n", myCandidate.x, myCandidate.y, myCandidate.z, leftNeighbor.x, leftNeighbor.y, leftNeighbor.z);
            myCandidate.x = UINT_MAX;
            myCandidate.y = UINT_MAX;
            myCandidate.z = UINT_MAX;
            break;
        }
    }

    candidates[globalId] = myCandidate;
}

__host__
extern "C" void cudaLaunchInvalidateOverlappingBlocksKernel(
    uint3* candidates,
    const int numCandidatesToFind,
    const int updatePhaseBatchSize
) {

    // Check 'updatePhaseBatchSize' blocks at a time and invalidate any that are overlapping
    // During the update phase the blocks will be processed in batches of this size, and any overlapping blocks in the same batch can cause divergence
    int numBlocks = numCandidatesToFind / updatePhaseBatchSize + (numCandidatesToFind % updatePhaseBatchSize == 0 ? 0 : 1);
    cuda_invalidateOverlappingBlocks << < numBlocks, updatePhaseBatchSize, updatePhaseBatchSize * sizeof(uint3) >> > (candidates, numCandidatesToFind, BLOCK_SIZE);
    hipDeviceSynchronize();
    cudaCheckExecution();
}


__host__
extern "C" void cudaInitializeRNGStates(hiprandState** rngStateOnGPU) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // setup execution parameters
    int threadsPerBlock = NUM_WORKERS;
    int maxConcurrentBlocks = deviceProperties.multiProcessorCount * 3; //TODO: Calculate this based on GPU max for # blocks
    int numThreads = maxConcurrentBlocks * threadsPerBlock;

    cudaCheckSuccess(hipMalloc(rngStateOnGPU, sizeof(hiprandState) * numThreads));
    cuda_InitCurandState << < maxConcurrentBlocks, threadsPerBlock >> > (*rngStateOnGPU);
    hipDeviceSynchronize();
    cudaCheckExecution();
}

__host__
extern "C" void cudaLaunchSolveDisplacementKernel(
    volatile Vertex* vertices,
    REAL* matConfigEquations,
    REAL* residualVolume,
    hiprandState* rngStateOnGPU,
    uint3* blockOrigins,
    const int numBlockOrigins,
    const uint3 solutionDims
) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // Blocks are divided into warps starting with x, then y, then z
    dim3 threadsPerBlock = { 32, 3, NUM_WORKERS };
    int maxConcurrentBlocks = deviceProperties.multiProcessorCount * 3; //TODO: Calculate this based on GPU max for # blocks
    int numIterations = std::max(numBlockOrigins / maxConcurrentBlocks, 1);

    cudaLaunchInvalidateOverlappingBlocksKernel(blockOrigins, numBlockOrigins, maxConcurrentBlocks);

#ifdef OUTPUT_NUM_FAILED_BLOCKS
    int numFailedBlocks = 0;
    for (int i = 0; i < numBlockOrigins; i++) {
        if (blockOrigins[i].x > solutionDims.x) {
            numFailedBlocks++;
        }
    }
    float percent = (static_cast<float>(numFailedBlocks) / numBlockOrigins) * 100);
    std::cout << numFailedBlocks << " of " << numBlockOrigins << " blocks overlapped (" << percent << "%)" << std::endl;
#endif

    // process all blocks in batches of maxConcurrentBlocks
    for (int i = 0; i < numIterations; i++) {
        uint3* currentBlockOrigins = &blockOrigins[i * maxConcurrentBlocks];
        int numBlocks = std::min(numBlockOrigins - i*maxConcurrentBlocks, maxConcurrentBlocks);

        cuda_SolveDisplacement << < numBlocks, threadsPerBlock >> >(vertices, matConfigEquations, residualVolume, solutionDims, currentBlockOrigins, rngStateOnGPU);
        hipDeviceSynchronize();
        cudaCheckExecution();
    }

}


