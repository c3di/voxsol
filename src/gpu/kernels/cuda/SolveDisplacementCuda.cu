#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <algorithm>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>
#include <>
#include "solution/Vertex.h"
#include "solution/samplers/BlockSampler.h"
#include "gpu/sampling/ResidualVolume.h"
#include "gpu/CudaCommonFunctions.h"
#include "gpu/GPUParameters.h"

#define MATRIX_ENTRY(rhsMatricesStartPointer, matrixIndex, row, col) rhsMatricesStartPointer[matrixIndex*9 + col*3 + row] //row*27*3 + col*27 + matrixIndex

//#define DYN_ADJUSTMENT_MAX 0.01f

__constant__ uint3 c_solutionDimensions;
__constant__ uint3 c_residualDimensions;


__device__ bool isInsideSolution(const uint3 coord) {
    return coord.x < c_solutionDimensions.x && coord.y < c_solutionDimensions.y && coord.z < c_solutionDimensions.z;
}

__device__ void buildRHSVectorForVertex(
    REAL rhsVec[27][3],
    Vertex localVertices[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2],
    const REAL* matrices,
    const char3& localCenterCoord
) {
    // We want to keep a full warp dedicated to each worker, but we only need enough threads for the 27 neighbors (minus the center vertex)
    const bool threadIsActive = threadIdx.x < 27 && threadIdx.x != CENTER_VERTEX_INDEX;
    unsigned activeThreadMask = __ballot_sync(__activemask(), threadIsActive);

    if (threadIsActive) {
        REAL rhsEntry[3] = { 0,0,0 };

        // Get coords of neighbor that this thread is responsible for, relative to the center vertex, in the 3x3x3 local problem
        const char localNeighborCoordX = (localCenterCoord.x + threadIdx.x % 3) - 1;
        const char localNeighborCoordY = (localCenterCoord.y + (threadIdx.x / 3) % 3) - 1;
        const char localNeighborCoordZ = (localCenterCoord.z + threadIdx.x / 9) - 1;
		
        const REAL nx = localVertices[localNeighborCoordZ][localNeighborCoordY][localNeighborCoordX].x;
        const REAL ny = localVertices[localNeighborCoordZ][localNeighborCoordY][localNeighborCoordX].y;
        const REAL nz = localVertices[localNeighborCoordZ][localNeighborCoordY][localNeighborCoordX].z;

        rhsEntry[0] = MATRIX_ENTRY(matrices, threadIdx.x, 0, 0) * nx + MATRIX_ENTRY(matrices, threadIdx.x, 0, 1) * ny + MATRIX_ENTRY(matrices, threadIdx.x, 0, 2) * nz;
        rhsEntry[1] = MATRIX_ENTRY(matrices, threadIdx.x, 1, 0) * nx + MATRIX_ENTRY(matrices, threadIdx.x, 1, 1) * ny + MATRIX_ENTRY(matrices, threadIdx.x, 1, 2) * nz;
        rhsEntry[2] = MATRIX_ENTRY(matrices, threadIdx.x, 2, 0) * nx + MATRIX_ENTRY(matrices, threadIdx.x, 2, 1) * ny + MATRIX_ENTRY(matrices, threadIdx.x, 2, 2) * nz;

        for (int offset = 16; offset > 0; offset /= 2) {
            rhsEntry[0] += __shfl_down_sync(activeThreadMask, rhsEntry[0], offset);
            rhsEntry[1] += __shfl_down_sync(activeThreadMask, rhsEntry[1], offset);
            rhsEntry[2] += __shfl_down_sync(activeThreadMask, rhsEntry[2], offset);
        }

        if (threadIdx.x == 0) {
            // Result of the shuffle reduction is stored in thread 0's variable
            rhsVec[threadIdx.y][0] = rhsEntry[0];
            rhsVec[threadIdx.y][1] = rhsEntry[1];
            rhsVec[threadIdx.y][2] = rhsEntry[2];
        }
    }
}

__device__ const REAL* getPointerToMatricesForVertexGlobal(Vertex* vertex, const REAL* matConfigEquations) {
    unsigned int equationIndex = static_cast<unsigned int>(vertex->materialConfigId) * (EQUATION_ENTRY_SIZE);
    return &matConfigEquations[equationIndex];
}

__device__ void updateVertex(Vertex* vertexToUpdate, REAL rhsVec[27][3], const REAL* matrices) {
    // Choose exactly 3 threads in the same warp to sum up the 3 RHS components and solve the system
	unsigned mask = __ballot_sync(__activemask(), threadIdx.x < 3);
    if (threadIdx.x < 3) {
        const char rhsComponentIndex = threadIdx.x;
        const char workerIndex = threadIdx.y;

        // Move to right side of equation and apply Neumann stress
        rhsVec[workerIndex][rhsComponentIndex] = -rhsVec[workerIndex][rhsComponentIndex] + matrices[NEUMANN_OFFSET + rhsComponentIndex];

        __syncwarp(mask);

        REAL newDisplacement = 0;
        newDisplacement += MATRIX_ENTRY(matrices, CENTER_VERTEX_INDEX, 0, rhsComponentIndex) * rhsVec[workerIndex][0];
        newDisplacement += MATRIX_ENTRY(matrices, CENTER_VERTEX_INDEX, 1, rhsComponentIndex) * rhsVec[workerIndex][1];
        newDisplacement += MATRIX_ENTRY(matrices, CENTER_VERTEX_INDEX, 2, rhsComponentIndex) * rhsVec[workerIndex][2];

        if (rhsComponentIndex == 0) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->x - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->x = newDisplacement;
        }
        if (rhsComponentIndex == 1) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->y - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->y = newDisplacement;
        }
        if (rhsComponentIndex == 2) {
#ifdef DYN_ADJUSTMENT_MAX
            if (abs(vertexToUpdate->z - newDisplacement) > DYN_ADJUSTMENT_MAX) {
                // Perform dynamical adjustment, discarding any displacement deltas that are larger than the epsilon defined in DYN_ADJUSTMENT_MAX
                // this is to prevent occasional large errors caused by race conditions. Smaller errors are corrected over time by the stochastic updates
                // of surrounding vertices
#ifdef OUTPUT_BAD_DISPLACEMENTS
                printf("Bad adjustment: %f diff for thread %i in block %i and bucket %i\n", newDisplacement, threadIdx.x, blockIdx.x, threadIdx.x / (blockDim.x / 2));
#endif
                return;
            }
#endif
            vertexToUpdate->z = newDisplacement;
        }

    }
}

__device__ void getUpdateCoordForThread(unsigned char subsetIndex, unsigned char vertexIndex, char3* updateCoord) {
    updateCoord->z = (vertexIndex / 9) * 2;
    updateCoord->y = ((vertexIndex / 3) % 3) * 2;
    updateCoord->x = (vertexIndex % 3) * 2;

    //+1 to account for the 1 vertex border around the update region
    updateCoord->x = updateCoord->x + subsetIndex % 2 + 1;
    updateCoord->y = updateCoord->y + (subsetIndex / 2) % 2 + 1;
    updateCoord->z = updateCoord->z + (subsetIndex / 4) % 2 + 1;
}


__device__ void updateVerticesInRegion(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    const REAL* matConfigEquations
) {
    __shared__ REAL rhsVec[27][3];

    // The local block has a 1-vertex border, so the valid update region actually starts at 1,1,1. This is taken into account below
    char3 localCoord = { 0,0,0 };
    unsigned char subsetIndex = 0;
    unsigned char vertexOffset = 0;

    // There are 9 workers but 27 vertices in each subset, so each subset needs to be divided into 3 sub-subsets
    for (int i = 0; i < UPDATES_PER_VERTEX * 3 * 8; i++) {
        getUpdateCoordForThread(subsetIndex, blockDim.y * vertexOffset + threadIdx.y, &localCoord);

        Vertex* vertexToUpdate = &localVertices[localCoord.z][localCoord.y][localCoord.x];
        const REAL* matrices = getPointerToMatricesForVertexGlobal(vertexToUpdate, matConfigEquations);
        buildRHSVectorForVertex(rhsVec, localVertices, matrices, localCoord);

        updateVertex(vertexToUpdate, rhsVec, matrices);

        vertexOffset = vertexOffset + 1;
        if (vertexOffset > 2) {
            vertexOffset = 0;
            subsetIndex = (subsetIndex + 1) % 8;
        }
    }
}

__device__
void copyVerticesFromGlobalToShared(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    volatile Vertex* verticesOnGPU,
    const uint3 blockOriginCoord
) {
    const int blockSizeWithBorder = BLOCK_SIZE + 2;
    const int numThreadsNeeded = blockSizeWithBorder * blockSizeWithBorder; //each thread will copy over a given x,y for all z ("top down")
    int threadIdx_1D = threadIdx.y * 32 + threadIdx.x;

    // Choose the first numThreadsNeeded threads to copy over the vertices
#pragma unroll
    for (unsigned char i = 0; i < 2; i++) {
        if (threadIdx_1D < numThreadsNeeded) {
            const char localCoordX = threadIdx_1D % blockSizeWithBorder;
            const char localCoordY = threadIdx_1D / blockSizeWithBorder;
            char localCoordZ = 0;

            for (int z = 0; z < BLOCK_SIZE + 2; z++) {
                localCoordZ = z;
                const uint3 globalCoord = { blockOriginCoord.x + localCoordX - 1, blockOriginCoord.y + localCoordY - 1, blockOriginCoord.z + z - 1 }; //-1 to account for border at both ends
                Vertex* local = &localVertices[localCoordZ][localCoordY][localCoordX];
                local->x = 0;
                local->y = 0;
                local->z = 0;
                local->materialConfigId = 0;

                if (isInsideSolution(globalCoord)) {
                    const int globalIndex = c_solutionDimensions.y*c_solutionDimensions.x*globalCoord.z + c_solutionDimensions.x*globalCoord.y + globalCoord.x;

                    //Turns out it's easier to copy the values manually than to get CUDA to play nice with a volatile struct assignment
                    volatile Vertex* global = &verticesOnGPU[globalIndex];
                    local->x = global->x;
                    local->y = global->y;
                    local->z = global->z;
                    local->materialConfigId = global->materialConfigId;
                }
            }
        }

        // There are less active threads than there are vertices to copy, so we need a second pass for the rest
        threadIdx_1D = blockDim.y * 32 + blockDim.x;
        threadIdx_1D = threadIdx_1D + threadIdx.y * 32 + threadIdx.x;
    }
}

__device__
void copyVerticesFromSharedToGlobalAndUpdateResiduals(
    Vertex localVertices[BLOCK_SIZE + 2][BLOCK_SIZE + 2][BLOCK_SIZE + 2],
    volatile Vertex* verticesOnGPU,
    const uint3 blockOriginCoord,
    REAL* residualVolume
) {
    const int numThreadsNeeded = BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE; //each thread will copy over one vertex in the inner block (without the border)
    int threadIdx_1D = threadIdx.y * 32 + threadIdx.x;

#pragma unroll
    for (unsigned char i = 0; i < 2; i++) {
        if (threadIdx_1D < numThreadsNeeded) {
            const char localCoordZ = 1 + threadIdx_1D / (BLOCK_SIZE * BLOCK_SIZE);
            const char localCoordY = 1 + (threadIdx_1D / BLOCK_SIZE) % BLOCK_SIZE;
            const char localCoordX = 1 + threadIdx_1D % BLOCK_SIZE;

            uint3 globalCoord = { 0,0,0 };
            globalCoord.z += localCoordZ + blockOriginCoord.z - 1;
            globalCoord.y += localCoordY + blockOriginCoord.y - 1;
            globalCoord.x += localCoordX + blockOriginCoord.x - 1;

            if (isInsideSolution(globalCoord)) {
                int globalIndex = c_solutionDimensions.y*c_solutionDimensions.x*globalCoord.z + c_solutionDimensions.x*globalCoord.y + globalCoord.x;
                const Vertex* local = &localVertices[localCoordZ][localCoordY][localCoordX];
                volatile Vertex* global = &verticesOnGPU[globalIndex];

                // First set residual to 0 for all updated vertices, then set the outer edge of vertices to the actual residual so future update blocks will
                // be placed near the edges of the current block, where the vertices are no longer in equilibrium.
                /*int residualIndex = (globalCoord.z + 1) / 2 * c_residualDimensions.y * c_residualDimensions.x + (globalCoord.y + 1) / 2 * c_residualDimensions.x + (globalCoord.x + 1) / 2;
                residualVolume[residualIndex] = asREAL(0.0);
                REAL residual = abs(global->x - local->x) + abs(global->y - local->y) + abs(global->z - local->z);
                if (localCoordZ == 1 || localCoordZ == BLOCK_SIZE) {
                    residualVolume[residualIndex] = residual;
                }
                if (localCoordY == 1 || localCoordY == BLOCK_SIZE) {
                    residualVolume[residualIndex] = residual;
                }
                if (localCoordX == 1 || localCoordX == BLOCK_SIZE) {
                    residualVolume[residualIndex] = residual;
                }*/

                global->x = local->x;
                global->y = local->y;
                global->z = local->z;

#ifdef OUTPUT_NAN_DISPLACEMENTS
                if (isnan(localVertices[localCoordZ][localCoordY][localCoordX].x)) {
                    printf("NAN encountered for block %i coord %u %u %u \n", blockIdx.x, localCoord.x, localCoord.y, localCoord.z);
                }
#endif
            }
        }

        // There are less active threads than there are vertices to copy, so we need a second pass for the rest
        threadIdx_1D = blockDim.y * 32 + blockDim.x;
        threadIdx_1D = threadIdx_1D + threadIdx.y * 32 + threadIdx.x;
    }
}

__global__
void cuda_SolveDisplacement(
    volatile Vertex* verticesOnGPU,
    REAL* matConfigEquations,
    const uint3* blockOrigins,
    REAL* residualVolume
) {
    const uint3 blockOriginCoord = blockOrigins[blockIdx.x];
    if (blockOriginCoord.x >= c_solutionDimensions.x || blockOriginCoord.y >= c_solutionDimensions.y || blockOriginCoord.z >= c_solutionDimensions.z) {
        // Some blocks may have been set to an invalid value during the importance sampling phase if they overlap with some other block, these
        // should not be processed
        return;
    }

    __shared__ Vertex localVertices[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
    
    copyVerticesFromGlobalToShared(localVertices, verticesOnGPU, blockOriginCoord);

    __syncthreads();

    updateVerticesInRegion(localVertices, matConfigEquations);
    
    __syncthreads();

    copyVerticesFromSharedToGlobalAndUpdateResiduals(localVertices, verticesOnGPU, blockOriginCoord, residualVolume); 

}

__global__
void cuda_invalidateOverlappingBlocks(uint3* candidates, const int numberOfCandidates, const unsigned int updateRegionSize) {
    extern __shared__ uint3 batch[];
    const int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalId >= numberOfCandidates) {
        return;
    }
    int localId = threadIdx.x;
    uint3 myCandidate = candidates[globalId];
    batch[localId] = myCandidate;

    __syncthreads();

    // Walk through the candidates toward the left
    while (localId > 0) {
        localId -= 1;
        const uint3 leftNeighbor = batch[localId];
        // Check for cube intersection, if any condition is true the two rectangular regions cannot intersect
        bool doesNotIntersect = false;
        doesNotIntersect = doesNotIntersect || leftNeighbor.x >= myCandidate.x + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.x + updateRegionSize <= myCandidate.x;
        doesNotIntersect = doesNotIntersect || leftNeighbor.z + updateRegionSize <= myCandidate.z;
        doesNotIntersect = doesNotIntersect || leftNeighbor.z >= myCandidate.z + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.y >= myCandidate.y + updateRegionSize;
        doesNotIntersect = doesNotIntersect || leftNeighbor.y + updateRegionSize <= myCandidate.y;
        if (!doesNotIntersect) {
            // Invalidate this block, it will later be skipped in the update phase since it lies outside the solution by definition of max_uint
            myCandidate.x = UINT_MAX;
            myCandidate.y = UINT_MAX;
            myCandidate.z = UINT_MAX;
            break;
        }
    }

    candidates[globalId] = myCandidate;
}

__host__
extern "C" void cudaLaunchInvalidateOverlappingBlocksKernel(
    uint3* candidates,
    const int numCandidatesToFind,
    const int updatePhaseBatchSize
) {

    // Check 'updatePhaseBatchSize' blocks at a time and invalidate any that are overlapping
    // During the update phase the blocks will be processed in batches of this size, and any overlapping blocks in the same batch can cause divergence
    int numBlocks = numCandidatesToFind / updatePhaseBatchSize + (numCandidatesToFind % updatePhaseBatchSize == 0 ? 0 : 1);
    cuda_invalidateOverlappingBlocks << < numBlocks, updatePhaseBatchSize, updatePhaseBatchSize * sizeof(uint3) >> > (candidates, numCandidatesToFind, BLOCK_SIZE);
    hipDeviceSynchronize();
    cudaCheckExecution();
}

__host__
extern "C" void cudaLaunchSolveDisplacementKernel(
    volatile Vertex* vertices,
    REAL* matConfigEquations,
    REAL* residualVolume,
    uint3* blockOrigins,
    const int numBlockOrigins,
    const uint3 solutionDims
) {
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    // Blocks are divided into warps starting with x, then y, then z
    dim3 threadsPerBlock = { 32, 9, 1 };
    int maxConcurrentBlocks = deviceProperties.multiProcessorCount * 8; //TODO: Calculate this based on GPU max for # blocks
    int numIterations = std::max(numBlockOrigins / maxConcurrentBlocks, 1);

    cudaLaunchInvalidateOverlappingBlocksKernel(blockOrigins, numBlockOrigins, maxConcurrentBlocks);

#ifdef OUTPUT_NUM_FAILED_BLOCKS
    int numFailedBlocks = 0;
    for (int i = 0; i < numBlockOrigins; i++) {
        if (blockOrigins[i].x > solutionDims.x) {
            numFailedBlocks++;
        }
    }
    float percent = (static_cast<float>(numFailedBlocks) / numBlockOrigins) * 100;
    std::cout << numFailedBlocks << " of " << numBlockOrigins << " blocks overlapped (" << percent << "%)" << std::endl;
#endif
    hipMemcpyToSymbol(HIP_SYMBOL(c_solutionDimensions), &solutionDims, sizeof(uint3));
    const uint3 residualDims = {(solutionDims.x+1) / 2, (solutionDims.y+1) / 2, (solutionDims.z+1) / 2};
    hipMemcpyToSymbol(HIP_SYMBOL(c_residualDimensions), &residualDims, sizeof(uint3));

    // process all blocks in batches of maxConcurrentBlocks
    for (int i = 0; i < numIterations; i++) {
        uint3* currentBlockOrigins = &blockOrigins[i * maxConcurrentBlocks];
        int numBlocks = std::min(numBlockOrigins - i*maxConcurrentBlocks, maxConcurrentBlocks);

        cuda_SolveDisplacement << < numBlocks, threadsPerBlock >> >(vertices, matConfigEquations, currentBlockOrigins, residualVolume);
        hipDeviceSynchronize();
        cudaCheckExecution();
    }

}


